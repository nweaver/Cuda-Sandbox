#include "hip/hip_runtime.h"
#include "matrix_mul.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>

Matrix<float> cudamul(Matrix<float> &a, Matrix<float> &b)
{
    return Matrix<float>();
}

__global__ void transpose(float *out, float *in, size_t size, size_t pitch)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    size_t x = tid % pitch;
    size_t y = tid / pitch;
    if (x < size && y < size)
    {
        out[x * pitch + y] = in[x + pitch * y];
    }
}

void checkResult(hipError_t error)
{
    if (error != hipSuccess)
    {
        std::cout << "CUDA error: " << hipGetErrorName(error) << "\n";

        std::cout << "CUDA error: " << hipGetErrorString(error) << "\n";
    }
}

float *cudaTransposeInternal(Matrix<float> &bin, size_t &pitch) {
    float *csource;
    float *cdest;
    auto result = hipMallocPitch(&csource,
                                  &pitch, bin._size * sizeof(float), bin._size);
    checkResult(result);

    result = hipMemcpy2D(csource, pitch, (void *)bin._data, bin._size * sizeof(float),
                          bin._size * sizeof(float), bin._size, hipMemcpyHostToDevice);
    checkResult(result);

    assert(result == hipSuccess);

    result = hipMallocPitch(&cdest,
                             &pitch, bin._size * sizeof(float), bin._size);
    checkResult(result);

    assert(result == hipSuccess);

    int block_size = 256;
    int grid_size = ((bin._size * bin._size + block_size) / block_size);

    transpose<<<grid_size, block_size>>>(cdest, csource, bin._size, pitch / sizeof(float));
    hipFree(csource);
    assert(result == hipSuccess);
    return cdest;
}


Matrix<float> cudaTranspose(Matrix<float> &bin)
{
    size_t pitch;
    float *cdest = cudaTransposeInternal(bin, pitch);

    Matrix<float> d;
    d._data = (float *)malloc(sizeof(float) * bin._size * bin._size);
    d._size = bin._size;
    auto result = hipMemcpy2D(d._data, bin._size * sizeof(float), cdest, pitch, bin._size * sizeof(float), bin._size,
                          hipMemcpyDeviceToHost);
    checkResult(result);

    assert(result == hipSuccess);

    result = hipFree(cdest);
    checkResult(result);

    assert(result == hipSuccess);

    return d;
}