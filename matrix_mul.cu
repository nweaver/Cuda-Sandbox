#include "hip/hip_runtime.h"
#include "matrix_mul.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>

Matrix<float> cudamul(Matrix<float> &a, Matrix<float> &b)
{
    return Matrix<float>();
}

__global__ void transpose(float *out, float *in, size_t size, size_t pitch)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    size_t x = tid % pitch;
    size_t y = tid / pitch;
    if (x < size && y < size)
    {
        out[x * pitch + y] = in[x + pitch * y];
    }
}

void checkResult(hipError_t error){
    if (error != hipSuccess){
                std::cout << "CUDA error: " << hipGetErrorName(error) << "\n";

        std::cout << "CUDA error: " << hipGetErrorString(error) << "\n";
    }
}

Matrix<float> cudaTranspose(Matrix<float> &bin)
{
    float *csource;
    float *cdest;
    size_t pitch;
    auto result = hipMallocPitch(&csource,
                                  &pitch, bin._size, bin._size);
checkResult(result);

    result = hipMemcpy2D(csource, pitch, (void *)bin._data, bin._size,
                          bin._size, bin._size, hipMemcpyHostToDevice);
                          checkResult(result);

    assert(result == hipSuccess);

    result = hipMallocPitch(&cdest,
                             &pitch, bin._size, bin._size);
                             checkResult(result);

    assert(result == hipSuccess);

    int block_size = 256;
    int grid_size = ((bin._size * bin._size + block_size) / block_size);

    transpose<<<grid_size, block_size>>>(cdest, csource, bin._size, pitch);

    Matrix<float> d;
    d._data = (float *)malloc(sizeof(float) * bin._size * bin._size);
    d._size = bin._size;
    result = hipMemcpy2D(d._data, bin._size, cdest, pitch, bin._size, bin._size,
                          hipMemcpyDeviceToHost);
                          checkResult(result);

    
    assert(result == hipSuccess);
    result = hipFree(csource);
    checkResult(result);

    assert(result == hipSuccess);

    result = hipFree(cdest);
    checkResult(result);

    assert(result == hipSuccess);

    return d;
}