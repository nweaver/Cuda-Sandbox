#include "hip/hip_runtime.h"
#include "matrix_mul.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>

float *cudaTransposeInternal(Matrix<float> &bin, size_t &pitch);

void checkResult(hipError_t error)
{
    if (error != hipSuccess)
    {
        std::cout << "CUDA error: " << hipGetErrorName(error) << "\n";

        std::cout << "CUDA error: " << hipGetErrorString(error) << "\n";
    }
}
__global__ void matmul(float *out, float *a, float *b, size_t size, size_t pitch)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    size_t x = tid % pitch;
    size_t y = tid / pitch;
    if (x < size && y < size)
    {
        float c = 0.0;
        int i;
        for (i = 0; i < size; ++i)
        {
            c += a[x * pitch + i] + b[y * pitch + i];
        }
        out[x + pitch * y] = c;
    }
}

Matrix<float> cudamul(Matrix<float> &a, Matrix<float> &b)
{
    size_t pitch;
    float *bin = cudaTransposeInternal(b, pitch);
    float *ain;
    float *dout;
    auto result = hipMallocPitch(&ain,
                                  &pitch, a._size * sizeof(float), a._size);
    checkResult(result);
    result = hipMemcpy2D(ain, pitch, (void *)a._data, a._size * sizeof(float),
                          a._size * sizeof(float), a._size, hipMemcpyHostToDevice);
    checkResult(result);
    result = hipMallocPitch(&dout,
                             &pitch, a._size * sizeof(float), a._size);
    checkResult(result);

    assert(result == hipSuccess);

    int block_size = 256;
    int grid_size = ((a._size * a._size + block_size) / block_size);

    matmul<<<grid_size, block_size>>>(dout, ain, bin, b._size, pitch / sizeof(float));
    hipFree(ain);
    hipFree(bin);

    Matrix<float> d;
    d._data = (float *)malloc(sizeof(float) * b._size * b._size);
    d._size = a._size;
    result = hipMemcpy2D(d._data, a._size * sizeof(float), dout, pitch, a._size * sizeof(float), a._size,
                          hipMemcpyDeviceToHost);
    checkResult(result);

    assert(result == hipSuccess);

    result = hipFree(dout);
    checkResult(result);

    assert(result == hipSuccess);
    return d;
}

__global__ void transpose(float *out, float *in, size_t size, size_t pitch)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    size_t x = tid % pitch;
    size_t y = tid / pitch;
    if (x < size && y < size)
    {
        out[x * pitch + y] = in[x + pitch * y];
    }
}

float *cudaTransposeInternal(Matrix<float> &bin, size_t &pitch)
{
    float *csource;
    float *cdest;
    auto result = hipMallocPitch(&csource,
                                  &pitch, bin._size * sizeof(float), bin._size);
    checkResult(result);

    result = hipMemcpy2D(csource, pitch, (void *)bin._data, bin._size * sizeof(float),
                          bin._size * sizeof(float), bin._size, hipMemcpyHostToDevice);
    checkResult(result);

    assert(result == hipSuccess);

    result = hipMallocPitch(&cdest,
                             &pitch, bin._size * sizeof(float), bin._size);
    checkResult(result);

    assert(result == hipSuccess);

    int block_size = 256;
    int grid_size = ((bin._size * bin._size + block_size) / block_size);

    transpose<<<grid_size, block_size>>>(cdest, csource, bin._size, pitch / sizeof(float));
    hipFree(csource);
    assert(result == hipSuccess);
    return cdest;
}

Matrix<float> cudaTranspose(Matrix<float> &bin)
{
    size_t pitch;
    float *cdest = cudaTransposeInternal(bin, pitch);

    Matrix<float> d;
    d._data = (float *)malloc(sizeof(float) * bin._size * bin._size);
    d._size = bin._size;
    auto result = hipMemcpy2D(d._data, bin._size * sizeof(float), cdest, pitch, bin._size * sizeof(float), bin._size,
                               hipMemcpyDeviceToHost);
    checkResult(result);

    assert(result == hipSuccess);

    result = hipFree(cdest);
    checkResult(result);

    assert(result == hipSuccess);
    return d;
}